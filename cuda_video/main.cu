#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>

#include <iostream>
#include <algorithm>

__device__ unsigned char my_min(unsigned char a, unsigned char b)
{
	return a < b ? a : b;
}

__global__ void ornekKernel(uchar3* src, uchar3* dst, int cols, int rows, const int factor)
{
	int x = blockIdx.x * (blockDim.x + threadIdx.x);
	int y = blockIdx.y * (blockDim.y + threadIdx.y);

	if (x < cols && y < rows)
	{
		int index = y * cols + x;

		dst[index].x = my_min(src[index].x * factor, (unsigned char)255);
		dst[index].y = my_min(src[index].y * factor, (unsigned char)255);
		dst[index].z = my_min(src[index].z * factor, (unsigned char)255);
		
	}
}



int main()
{
	cv::VideoCapture cap(0);

	if (!cap.isOpened())
	{
		std::cerr << "Kamera acilmadi" << std::endl;
		return -1;
	}

	while (true)
	{
		cv::Mat frame;
		cap >> frame;

		hipError_t err;
		uchar3* devSrc;
		uchar3* devDst;


		int cols = frame.cols;
		int rows = frame.rows;


		err = hipMalloc((void**)&devSrc, cols * rows * sizeof(uchar3));

		if (err != hipSuccess)
		{
			std::cerr << "Hata: " << hipGetErrorString(err) << std::endl;
			break;
		}


		err = hipMalloc((void**)&devDst, cols * rows * sizeof(uchar3));

		if (err != hipSuccess)
		{
			std::cerr << "Hata: " << hipGetErrorString(err) << std::endl;
			break;
		}

		hipMemcpy(devSrc, frame.data, cols * rows * sizeof(uchar3), hipMemcpyHostToDevice);

		dim3 threadsPerBlock(16, 16);
		dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,(rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

		ornekKernel <<< numBlocks, threadsPerBlock >> > (devSrc, devDst, cols, rows, 10);

		cv::Mat output(rows, cols, CV_8UC3);

		hipMemcpy(output.data, devDst, cols * rows * sizeof(uchar3), hipMemcpyDeviceToHost);

		cv::imshow("G�r�nt�", output);

		hipFree(devSrc);
		hipFree(devDst);


		char k = cv::waitKey(30);
		if (k == 27) // ASCII kodu 27 => ESC tu�u
		{
			break;
		}

	}
		return 0;

}